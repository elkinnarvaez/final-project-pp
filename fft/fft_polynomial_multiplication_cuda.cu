
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <tgmath.h>
#include <omp.h>
#include <time.h>
#include <math.h>

using namespace std;

using cd = complex<double>;
const double PI = acos(-1);


void print_vector_complex(const vector<cd> &a){
    int n = a.size();
	for (int i = 0; i < n; i++) 
		cout << a[i] << " ";
    cout << endl;
}

void print_vector_int(const vector<int> &a){
    int n = a.size();
	for (int i = 0; i < n; i++) 
		cout << a[i] << " ";
    cout << endl;
}

void fft_parallel(vector<cd> & a, bool invert) {
    int n = a.size();
    if (n == 1)
        return;
    vector<cd> a0(n / 2), a1(n / 2);

    // for (int i = 0; i < n/2; i++) {
    //     a0[i] = a[2*i];
    //     a1[i] = a[2*i+1];
    // }

    #pragma omp parallel sections
    {
        #pragma omp section
            fft_parallel(a0, invert);
        #pragma omp section
            fft_parallel(a1, invert);
    }

    // #pragma omp single nowait
    // {
    //     #pragma omp task
    //         fft_parallel(a0, invert);
    //     #pragma omp task
    //         fft_parallel(a1, invert);
    // }
    // #pragma omp taskwait

    double ang = 2 * PI / n * (invert ? -1 : 1);
    cd w(1), wn(cos(ang), sin(ang));
    for (int i = 0; 2*i < n; i++) {
        a[i] = a0[i] + w * a1[i];
        a[i + n/2] = a0[i] - w * a1[i];
        if (invert) {
            a[i] /= 2;
            a[i + n/2] /= 2;
        }
        w *= wn;
    }
}

vector<int> multiply_parallel(vector<int> const& a, vector<int> const& b) {
    vector<cd> fa(a.begin(), a.end()), fb(b.begin(), b.end());
    int n = 1;
    while (n < a.size() + b.size())
        n <<= 1;
    fa.resize(n);
    fb.resize(n);

    #pragma omp parallel sections
    {
        #pragma omp section
            fft_parallel(fa, false);
        #pragma omp section
            fft_parallel(fb, false);
    }
    
    for (int i = 0; i < n; i++)
        fa[i] *= fb[i];
    fft_parallel(fa, true);

    vector<int> ans(n);
    
    #pragma omp parallel for
    for (int i = 0; i < n; i++)
        ans[i] = round(fa[i].real());
    return ans;
}

void fft_sequential(vector<cd> & a, bool invert) {
    int n = a.size();
    if (n == 1)
        return;
    vector<cd> a0(n / 2), a1(n / 2);
    for (int i = 0; 2 * i < n; i++) {
        a0[i] = a[2*i];
        a1[i] = a[2*i+1];
    }
    fft_sequential(a0, invert);
    fft_sequential(a1, invert);

    double ang = 2 * PI / n * (invert ? -1 : 1);
    cd w(1), wn(cos(ang), sin(ang));
    for (int i = 0; 2 * i < n; i++) {
        a[i] = a0[i] + w * a1[i];
        a[i + n/2] = a0[i] - w * a1[i];
        if (invert) {
            a[i] /= 2;
            a[i + n/2] /= 2;
        }
        w *= wn;
    }
}

vector<int> multiply_sequential(vector<int> const& a, vector<int> const& b) {
    vector<cd> fa(a.begin(), a.end()), fb(b.begin(), b.end());
    int n = 1;
    while (n < a.size() + b.size())
        n <<= 1;
    fa.resize(n);
    fb.resize(n);

    fft_sequential(fa, false);
    fft_sequential(fb, false);
    for (int i = 0; i < n; i++)
        fa[i] *= fb[i];
    fft_sequential(fa, true);

    vector<int> ans(n);
    for (int i = 0; i < n; i++)
        ans[i] = round(fa[i].real());
    return ans;
}

void running_time_multiple_cases(){
    int N = (int)pow(2, 20);
    double start, end, elapsed;
    for(int n = 2; n <= N; n *= 2){
        vector<int> a(n), b(n);
        srand (time(NULL));
        for(int i = 0; i < n; i++){
            a[i] = rand() % 10 + 1;
            b[i] = rand() % 10 + 1;
        }
        start = omp_get_wtime();
        multiply_parallel(a, b);
        end = omp_get_wtime();
        elapsed = end - start;
        printf("%d %f\n", n, elapsed);
    }
}

void fft_test(){
 	vector<cd> a{-3, 4, 1, 5};
	fft_parallel(a, false);
    print_vector_complex(a);
    fft_parallel(a, true);
    print_vector_complex(a);   
}

void multiply_test(){
	vector<int> a = {5, 6, -3, 6, 8, 17, -9, 1};
    vector<int> b = {7, 4, 8, -3, 7};
    vector<int> ans = multiply_parallel(a, b);
    print_vector_int(ans);
}

int main(){
    running_time_multiple_cases();
    return 0;
}